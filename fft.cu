#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <sys/time.h>

#define INPUT_SIZE 5120

int main(){

	struct timeval start, end;
	hipError_t err;
	hipfftResult res;

	double *idata = (double *)malloc(INPUT_SIZE * sizeof(double));
	for(int i=0; i<INPUT_SIZE; i++){
		idata[i] = rand() / (double)RAND_MAX;
	}
	double *odata = (double *)malloc(2*INPUT_SIZE * sizeof(double));
	for(int i=0; i<2*INPUT_SIZE; i++){
		odata[i] = 0.0;
	}

	double *d_idata, *d_odata;
	err = hipMalloc((void **)&d_idata, INPUT_SIZE*sizeof(double));
	if(err != hipSuccess){
		printf("hipMalloc failed\n");
	}
	err = hipMalloc((void **)&d_odata, 2*INPUT_SIZE*sizeof(double));
	if(err != hipSuccess){
		printf("hipMalloc failed\n");
	}

	hipfftHandle plan;
	res = hipfftPlan1d(&plan, INPUT_SIZE, HIPFFT_R2C, 1);
	if(res != HIPFFT_SUCCESS){
		printf("hipfftPlan1d failed\n");
	}

	for(int i=0; i<10; i++){
        gettimeofday(&start, NULL);
	err = hipMemcpy(d_idata, idata, INPUT_SIZE*sizeof(double), hipMemcpyHostToDevice);
	if(err != hipSuccess){
		printf("hipMemcpyHostToDevice failed\n");
	}
        gettimeofday(&end, NULL);
	printf("hipMemcpyHostToDevice took %llu us \n", (end.tv_sec - start.tv_sec)*1000000 + (end.tv_usec - start.tv_usec));

        gettimeofday(&start, NULL);
	res = hipfftExecR2C(plan, (hipfftReal *)d_idata, (hipfftComplex *)d_odata);
	if(res != HIPFFT_SUCCESS){
		printf("hipfftExecR2C failed\n");
	}
	err = hipDeviceSynchronize();
	if(err != hipSuccess){
		printf("hipDeviceSynchronize failed\n");
	}
        gettimeofday(&end, NULL);
	printf("hipfftExecR2C took %llu us \n", (end.tv_sec - start.tv_sec)*1000000 + (end.tv_usec - start.tv_usec));

        gettimeofday(&start, NULL);
	err = hipMemcpy(odata, d_odata, 2*INPUT_SIZE*sizeof(double), hipMemcpyDeviceToHost);
	if(err != hipSuccess){
		printf("hipMemcpyDeviceToHost failed\n");
	}
        gettimeofday(&end, NULL);
	printf("hipMemcpyDeviceToHost took %llu us \n", (end.tv_sec - start.tv_sec)*1000000 + (end.tv_usec - start.tv_usec));
	}

	return 0;

}
