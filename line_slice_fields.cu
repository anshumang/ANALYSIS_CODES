
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_CHECK(cmd) {hipError_t error = cmd; if(error!=hipSuccess) std::cout << hipGetErrorString(error) << std::endl;}

__global__ void kernelLineSliceFields(hipPitchedPtr fieldE, hipPitchedPtr fieldB, float3 *sliceDataField, dim3 globalCellIdOffset, dim3 globalNrOfCells)
{
    dim3 superCellIdx;
    //superCellIdx(mapper.getSuperCellIndex(blockIdx))
    superCellIdx.x = blockIdx.x/16 + 1; 
    superCellIdx.y = blockIdx.y + 1; 
    superCellIdx.z = blockIdx.x%16 + 1; 

    __syncthreads();

    dim3 localCell, superCellSize;
    superCellSize.x = 8; superCellSize.y = 8; superCellSize.z = 4;
    localCell.x = superCellIdx.x * superCellSize.x + threadIdx.x;
    localCell.y = superCellIdx.y * superCellSize.y + threadIdx.y;
    localCell.z = superCellIdx.z * superCellSize.z + threadIdx.z;

    char *fieldEPtr = (char *)fieldE.ptr;
    size_t eSlicePitch = fieldE.pitch * fieldE.ysize;
    char *eSlice = fieldEPtr + localCell.z * eSlicePitch;
    float3 *eRow = (float3 *)eSlice + localCell.y * fieldE.pitch;
    float3 e = eRow[localCell.x];   

    char *fieldBPtr = (char *)fieldB.ptr;
    size_t bSlicePitch = fieldB.pitch * fieldB.ysize;
    char *bSlice = fieldBPtr + localCell.z * bSlicePitch;
    float3 *bRow = (float3 *)bSlice + localCell.y * fieldB.pitch;
    float3 b = bRow[localCell.x];   

    dim3 localCellWG;
    int guardingSuperCells = 1;
    localCellWG.x = localCell.x - superCellSize.x * guardingSuperCells; 
    localCellWG.y = localCell.y - superCellSize.y * guardingSuperCells; 
    localCellWG.z = localCell.z - superCellSize.z * guardingSuperCells; 

    dim3 globalCell;
    globalCell.x = localCellWG.x + globalCellIdOffset.x; 
    globalCell.y = localCellWG.y + globalCellIdOffset.y; 
    globalCell.z = localCellWG.z + globalCellIdOffset.z; 

    if(globalCell.x == globalNrOfCells.x /2)
	if(globalCell.z == globalNrOfCells.z /2)
	     sliceDataField[localCellWG.y] = e;

    __syncthreads();
}

int main(){
     hipExtent extent;
     hipPitchedPtr d_field_e, d_field_b;
     extent.width = 960 * sizeof(float3);
     extent.height = 80;
     extent.depth = 72;
     CUDA_CHECK(hipMalloc3D(&d_field_e, extent));//960 80 72
     CUDA_CHECK(hipMalloc3D(&d_field_b, extent));//960 80 72
     CUDA_CHECK(hipMemset3D(d_field_e, 0, extent));
     CUDA_CHECK(hipMemset3D(d_field_b, 0, extent));
     float3 *d_slice_data_field;
     size_t slice_data_field_pitch = 1;
     CUDA_CHECK(hipMallocPitch(&d_slice_data_field, &slice_data_field_pitch, 64 * sizeof(float3), 1));
     dim3 grid(128, 8, 1), block(8, 8, 4);
     dim3 global_cell_id_offset(0, 0, 0), global_nr_of_cells(128, 128, 128);
     for(int i=0; i<25; i++){ //-s 25 -lslice.period 1
     //wait_for();
     kernelLineSliceFields<<<grid, block>>>(d_field_e, d_field_b, d_slice_data_field, global_cell_id_offset, global_nr_of_cells);
     CUDA_CHECK(hipDeviceSynchronize());
     //release();
     }
     return 0;
}
